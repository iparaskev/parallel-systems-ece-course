
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <limits.h>
#include <sys/time.h>

#define stride 4

/* overload atomicAdd for double */
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

  #else
  static __inline__ __device__ double atomicAdd(double *address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    if (val==0.0)
      return __longlong_as_double(old);
    do {
      assumed = old;
      old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
  }
  #endif


// global variables
int rows, columns;
__device__ int dev_rows, dev_columns, dev_sparse_count = 0;
__device__ double dev_h, dev_norm;

// device functions 
__global__ void get_exp(double *x, double *y, double *w);
__global__ void get_exp_shared(double *x, double *y, double *w);
__global__ void multiply_arrays(double *x, double *w, double *y_new);
__global__ void get_sum_per_row(double *w, double *sums);
__global__ void mean_shift(double *y_new, double *sums_array, double *y, double *norms);
__global__ void compute_norms(double *norms, double *norm_per_block);

// local cpu functions
double *read_data(const char *name);
int check_results(double *y, char *name);
void write_results(char **argv, int iterations, double time);

double
now()
{
 struct timeval tv;
 gettimeofday(&tv, 0);
 return tv.tv_sec + tv.tv_usec / 1000000.0;
}

int main(int argc, char **argv)
{
  double *x, *y, h, epsilon, norm, *sums_array;
  double *dev_x, *dev_y, *dev_w, *dev_y_new, *dev_sums_array, *dev_norms;
  double *dev_blocks_norm; 
  int *dev_counter_per_row;

  if (argc < 6){
    fprintf(stderr, "it needs 5 arguments, which are: h iterations data validation_data dims\n");
    exit(1);
  }
  // initialize arrays at the host
  int iterations = atoi(argv[2]);
  columns = atoi(argv[5]);
  x = read_data(argv[3]);
  printf("row %d\n", rows);
  y = (double *) malloc(sizeof *y * rows * columns);
  memcpy(y, x, sizeof(double) * rows * columns);
  sums_array = (double *) malloc(sizeof *sums_array * rows);
  h = atoi(argv[1]);
  epsilon = 1e-4*h;
  
  hipError_t error;
  
  // parameters for reduction per block for the computation of norm
  int blocks_size = 512;
  int thread_size = blocks_size;
  double *blocks_norm = (double *) malloc(sizeof *blocks_norm * (blocks_size / 2));
  
  // initialize at the device
  error = hipMalloc((void **) &dev_x, rows * columns * sizeof(double));
  if (error)
    printf("malloc dev_x %s\n", hipGetErrorString(error));
  
  error = hipMalloc((void **) &dev_y, rows * columns * sizeof(double));
  if (error)
    printf("malloc dev_y %s\n", hipGetErrorString(error));
  
  error = hipMalloc((void **) &dev_y_new, rows * columns * sizeof(double));
  if (error)
    printf("malloc dev_y_new %s\n", hipGetErrorString(error));
  
  error = hipMalloc((void **) &dev_sums_array, rows * sizeof(double));
  if (error)
    printf("malloc dev_sums_array %s\n" ,hipGetErrorString(error));
  
  error = hipMalloc((void **) &dev_w, (rows * rows) * sizeof(double));
  
  // allocate max memory for sparse
  int paranom = 2;
  while (error){
    error = hipMalloc((void **) &dev_w, (rows * rows) / paranom++ * sizeof(double));
    if (paranom > 20){
      fprintf(stderr, "%s\n", "no memory");
      return 1;
    }
  }
  
  error = hipMalloc((void **) &dev_counter_per_row, rows * sizeof(int));
  if (error)
    printf("malloc dev_counter_per_row %s\n", hipGetErrorString(error));
  
  error = hipMalloc((void **) &dev_norms, blocks_size * thread_size * sizeof(double));
  if (error)
    printf("malloc dev_norms %s\n", hipGetErrorString(error));
  
  error = hipMalloc((void **) &dev_blocks_norm, (blocks_size / 2) * sizeof(double));
  if (error)
    printf("malloc dev_blocks_norm %s\n", hipGetErrorString(error));
  
  // copy initial arrays from host to device
  hipMemcpy(dev_x, x, rows * columns * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(dev_y, y, rows * columns * sizeof(double), hipMemcpyHostToDevice);

  hipMemcpyToSymbol(HIP_SYMBOL(dev_rows), &rows, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_columns), &columns, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(dev_h), &h, sizeof(double));
  
  norm = INT_MAX;
  double time_passed;
  double t_start = now();
  // for(int j = 0; j < iterations; j++){
  int j = 0;
  while(sqrt(norm) > epsilon){  
    norm = 0;
    
    // fill sparse array
    get_exp<<<256, 64>>>(dev_x, dev_y, dev_w);
    hipDeviceSynchronize();
    
    // zero new array for the sums
    memset(y, 0, rows * columns * sizeof(double));
    hipMemcpy(dev_y_new, y, rows * columns * sizeof(double), hipMemcpyHostToDevice);
   
    multiply_arrays<<<256, 64>>>(dev_x, dev_w, dev_y_new);
    hipDeviceSynchronize();
    
    // sum sparse
    memset(sums_array, 0, rows * sizeof(double));
    hipMemcpy(dev_sums_array, sums_array, rows * sizeof(double), hipMemcpyHostToDevice);
    get_sum_per_row<<<512, 512>>>(dev_w, dev_sums_array);
    hipDeviceSynchronize();

    // compute mean shift
    mean_shift<<<blocks_size, 512>>>(dev_y_new, dev_sums_array, dev_y, dev_norms);
    hipDeviceSynchronize();
    
    compute_norms<<<blocks_size / 2, 512>>>(dev_norms, dev_blocks_norm);
    hipDeviceSynchronize();
    hipMemcpy(blocks_norm, dev_blocks_norm, (blocks_size / 2) * sizeof(double), hipMemcpyDeviceToHost);
    for (int k = 0; k < (blocks_size/2); k++)
      norm += blocks_norm[k];
    
    printf("iteration %d error %f \n", j, sqrt(norm));

    j++;
  }
  
  time_passed = now() - t_start;
  printf("time passed %f\n", time_passed);

    
  hipMemcpy(y, dev_y, rows * columns * sizeof(double), hipMemcpyDeviceToHost);
  check_results(y, argv[4]);
  FILE *f = fopen("res.txt", "w");
  for (int i = 0; i < rows; i++){
    for (int j = 0; j < columns; j++)
      fprintf(f, "%f ", y[i * columns + j]);
    fprintf(f, "\n");
  }
  fclose(f);

  // clean up gpu
  hipFree((void *) &dev_x);
  hipFree((void *) &dev_y);
  hipFree((void *) &dev_w);
  hipFree((void *) &dev_sums_array);
  hipFree((void *) &dev_norms);
  hipFree((void *) &dev_blocks_norm);
  hipFree((void *) &dev_y_new);

  // clean up cpu
  free(x);
  free(y);
  free(sums_array);

  return 0;
}

__global__ void
multiply_arrays(double *x, double *w, double *y_new)
{
  double tmp = 0;
  
  int row, column;
  double dist;
  int global_thread_id = threadIdx.x + blockIdx.x * blockDim.x;

  // get the first element of one (row,column,dist) of sparse
  int w_tid = (global_thread_id) * 3;
  while (w_tid < 3 * dev_sparse_count){
    row = w[w_tid];
    column = w[w_tid + 1];
    dist = w[w_tid + 2];

    // mulitply distance with every element in the row
    for(int i = 0; i < dev_columns; i++){
      tmp = dist * x[column * dev_columns + i];
      atomicAdd(&y_new[row * dev_columns + i], tmp);
    }

    w_tid += blockDim.x * gridDim.x * 3;
  }
}

__global__ void
get_sum_per_row(double *w, double *sums)
{
  int global_thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  int row;
  double dist;

  int w_tid = (global_thread_id) * 3;
  while (w_tid < dev_sparse_count * 3){
    row = w[w_tid];
    dist = w[w_tid + 2];
    atomicAdd(&sums[row], dist);
    
    w_tid += blockDim.x * gridDim.x * 3;
  }
}

__global__ void
mean_shift(double *y_new, double *sums_array, double *y, double *norms)
{
  int y_tid = threadIdx.x + blockIdx.x * blockDim.x;
  int sum_tid = y_tid / dev_columns;
  double m_dif = 0;
  int norms_tid = y_tid;
  norms[norms_tid] = 0;
  dev_sparse_count  = 0;

  while (y_tid < (dev_rows * dev_columns)){
    y_new[y_tid] /= sums_array[sum_tid];
    m_dif = y_new[y_tid] - y[y_tid];
    norms[norms_tid] += pow(m_dif, 2);
    y[y_tid] = y_new[y_tid];

    y_tid += gridDim.x * blockDim.x;
    sum_tid = y_tid / dev_columns;
  }
}

__global__ void 
compute_norms(double *norms, double *norm_per_block)
{
  int n_tid = 2 * (threadIdx.x + blockIdx.x * blockDim.x);
  int i = 1;
  int initial_tid = n_tid / 2;
  int block_limit = gridDim.x * blockDim.x;

  int block_end = 2 * (blockIdx.x * blockDim.x + blockDim.x) - 1; 

  if (n_tid < (2 * block_limit)){
    
    while ( (i < (2 * blockDim.x)) && n_tid < block_end && 
           (n_tid + i) <= block_end){
      
      norms[n_tid] += norms[n_tid + i];
      /* update n_tid with respect to the start of the block
         double the offset of n_tid to kill half of the threads */
      n_tid = n_tid + i * 2 * (initial_tid - (blockIdx.x * blockDim.x));
      i *= 2;
      __syncthreads();
      
    }

    // element with threadid = 0 adds the sum 
    if (!((initial_tid) % blockDim.x))
      norm_per_block[blockIdx.x] = norms[n_tid];
    
  } 
}

__device__ double
compute_distance(double *y_i, double *x_j, double limit)
{
  double dist = 0, tmp;
  for (int i = 0; i < dev_columns; i++){
    tmp = y_i[i] - x_j[i];
    dist += tmp * tmp;
    if (dist > limit)
      return 0;
  }

  return exp( -dist / (2 * limit));
}

__global__ void
get_exp(double *x, double *y, double *w)
{
  double dist;
  double limit = dev_h * dev_h;
  
  int pos;

  long int global_thread_id = threadIdx.x + blockIdx.x * blockDim.x;
  long int y_tid = global_thread_id / (dev_rows / stride);
  long int x_tid = stride * (global_thread_id % (dev_rows / stride));
  
  while(y_tid < dev_rows && x_tid < (dev_rows)){
    
    for (int i = 0; (i < stride); i++){
      
      dist = compute_distance(&y[y_tid * dev_columns], &x[x_tid * dev_columns], limit);
      // insert elements to sparse
      if (dist){

        pos = atomicAdd(&dev_sparse_count, 1);
        
        w[pos * 3] = y_tid;
        w[pos * 3 + 1] = x_tid;
        w[pos * 3 + 2] = dist;
      }
      x_tid++;
    }
    global_thread_id += blockDim.x * gridDim.x;
      
    y_tid = global_thread_id / (dev_rows / stride);
    x_tid = stride * (global_thread_id % (dev_rows/stride));
  }
}

double *
read_data(const char *name)
{
  FILE *f;
  f = fopen(name, "rb");
  fseek(f, 0L, SEEK_END);
  int pos = ftell(f);
  fseek(f, 0L, SEEK_SET);

  int number_elements = pos / sizeof(double);
  double *x = (double *) malloc(sizeof *x * number_elements);
  fread(x, sizeof *x, number_elements, f);
  rows = number_elements / columns;
  fclose(f);

  return x;
}

int 
check_results(double *y, char *name)
{
  double *mat_res = read_data(name);
  double dist = 0;
  for (int i = 0; i < rows; i++){
    for (int j = 0; j < columns; j++){
      dist += pow(y[i * columns + j] - mat_res[i * columns + j], 2);
      // if (dist > 1)
          // printf("%f %f %f\n", y[i * columns + j], mat_res[i * columns + j], dist);
    }
  }
  printf("done dist %.10f \n", dist);
  return 0;
}

