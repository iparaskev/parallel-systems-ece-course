
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <limits.h>
#include <sys/time.h>

#define BLOCK_SIZE 16
#define stride 4
#define DIMS 2

/* overload atomicAdd for double */
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

#else
static __inline__ __device__ 
double atomicAdd(double *address, double val) 
{
		unsigned long long int* address_as_ull = (unsigned long long int*)address;
		unsigned long long int old = *address_as_ull, assumed;
		if (val==0.0)
			return __longlong_as_double(old);
		do {
			assumed = old;
			old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
		} while (assumed != old);
		return __longlong_as_double(old);
}
#endif


// global variable
int rows, columns;
__device__ int dev_rows, dev_columns, dev_sparse_count = 0;
__device__ double dev_h, dev_norm;

// device functions 
__global__ void get_exp_shared(double *x, double *y, double *w);
__global__ void multiply_arrays(double *x, double *w, double *y_new);
__global__ void get_sum_per_row(double *w, double *sums);
__global__ void mean_shift(double *y_new, double *sums_array, double *y, double *norms);
__global__ void compute_norms(double *norms, double *norm_per_block);

// local cpu functions
double *read_data(const char *name);
int check_results(double *y, char *name);
void write_results(char **argv, int iterations, double time);

double
now()
{
        struct timeval tv;
        gettimeofday(&tv, 0);
        return tv.tv_sec + tv.tv_usec / 1000000.0;
}

int 
main(int argc, char **argv)
{
        double *x, *y, h, epsilon, norm, *sums_array;
        double *dev_x, *dev_y, *dev_w, *dev_y_new, *dev_sums_array, *dev_norms;
        double *dev_blocks_norm; 

	    if (argc < 6){
	    	fprintf(stderr, "it needs 5 arguments, which are: h iterations data validation_data dims\n");
	    	exit(1);
	    }
	    // initialize arrays at the host
	    int iterations = atoi(argv[2]);
	    columns = atoi(argv[5]);
	    x = read_data(argv[3]);
	    y = (double *) malloc(sizeof *y * rows * columns);
	    memcpy(y, x, sizeof(double) * rows * columns);
	    sums_array = (double *) malloc(sizeof *sums_array * rows);
	    h = atoi(argv[1]);
	    epsilon = 1e-4*h;
	    
	    // parameters for reduction per block for the computation of norm
	    int blocks_size = 512;
	    int thread_size = blocks_size;
	    double *blocks_norm = (double *) malloc(sizeof *blocks_norm * (blocks_size / 2));
	    
	    hipError_t error;
	    // initialize at the device
	    error = hipMalloc((void **) &dev_x, rows * columns * sizeof(double));
	    if (error)
	    	printf("malloc dev_x %s\n", hipGetErrorString(error));
	    
	    error = hipMalloc((void **) &dev_y, rows * columns * sizeof(double));
	    if (error)
	    	printf("malloc dev_y %s\n", hipGetErrorString(error));
	    
	    error = hipMalloc((void **) &dev_y_new, rows * columns * sizeof(double));
	    if (error)
	    	printf("malloc dev_y_new %s\n", hipGetErrorString(error));
	    
	    error = hipMalloc((void **) &dev_sums_array, rows * sizeof(double));
	    if (error)
	    	printf("malloc dev_sums_array %s\n" ,hipGetErrorString(error));
	    
	    error = hipMalloc((void **) &dev_w, (rows * rows) * sizeof(double));
	    
	    // allocate max memory for sparse
	    int paranom = 2;
	    while (error){
	    	error = hipMalloc((void **) &dev_w, (rows * rows) / paranom++ * sizeof(double));
	    	if (paranom > 20){
	    		fprintf(stderr, "%s\n", "no memory");
	    		return 1;
	    	}
	    }
	    	
	    error = hipMalloc((void **) &dev_norms, blocks_size * thread_size * sizeof(double));
	    if (error)
	    	printf("malloc dev_norms %s\n", hipGetErrorString(error));
	    
	    error = hipMalloc((void **) &dev_blocks_norm, (blocks_size / 2) * sizeof(double));
	    if (error)
	    	printf("malloc dev_blocks_norm %s\n", hipGetErrorString(error));
	    
	    // copy initial arrays from host to device
	    hipMemcpy(dev_x, x, rows * columns * sizeof(double), hipMemcpyHostToDevice);
	    hipMemcpy(dev_y, y, rows * columns * sizeof(double), hipMemcpyHostToDevice);

	    hipMemcpyToSymbol(HIP_SYMBOL(dev_rows), &rows, sizeof(int));
	    hipMemcpyToSymbol(HIP_SYMBOL(dev_columns), &columns, sizeof(int));
	    hipMemcpyToSymbol(HIP_SYMBOL(dev_h), &h, sizeof(double));
	    
	    norm = INT_MAX;
	    double t_start = now(), time_passed;
	    
	    // for(int j = 0; j < iterations; j++){
	    int j = 0;
	    while(sqrt(norm) > epsilon){	
	    	norm = 0;
	    	
	    	get_exp_shared<<<256, BLOCK_SIZE * BLOCK_SIZE/stride>>>(dev_x, dev_y, dev_w);
	    	hipDeviceSynchronize();
	    	
	    	// zero new array
	    	memset(y, 0, rows * columns * sizeof(double));
	    	hipMemcpy(dev_y_new, y, rows * columns * sizeof(double), hipMemcpyHostToDevice);
	    	
	    	multiply_arrays<<<256, 64>>>(dev_x, dev_w, dev_y_new);
	    	hipDeviceSynchronize();
	    	
	    	// sum sparse
	    	memset(sums_array, 0, rows * sizeof(double));
	    	hipMemcpy(dev_sums_array, sums_array, rows * sizeof(double), hipMemcpyHostToDevice);
	    	get_sum_per_row<<<512, 512>>>(dev_w, dev_sums_array);
	    	hipDeviceSynchronize();

	    	mean_shift<<<blocks_size, 512>>>(dev_y_new, dev_sums_array, dev_y, dev_norms);
	    	hipDeviceSynchronize();
	    	
	    	compute_norms<<<blocks_size / 2, 512>>>(dev_norms, dev_blocks_norm);
	    	hipDeviceSynchronize();
	    	hipMemcpy(blocks_norm, dev_blocks_norm, (blocks_size / 2) * sizeof(double), hipMemcpyDeviceToHost);
	    	for (int k = 0; k < (blocks_size/2); k++)
	    		norm += blocks_norm[k];
	    	
	    	printf("iteration %d error %f \n", j, sqrt(norm));
	    	// break;
	    	j++;
	    }
	    time_passed = now() - t_start;
	    printf("time passed %f\n", now() - t_start);
	    hipMemcpy(y, dev_y, rows * columns * sizeof(double), hipMemcpyDeviceToHost);
	    check_results(y, argv[4]);

	    FILE *f = fopen("res.txt", "w");
	    for (int i = 0; i < rows; i++){
	    	for (int j = 0; j < columns; j++)
	    		fprintf(f, "%f ", y[i * columns + j]);
	    	fprintf(f, "\n");
	    }
	    fclose(f);

	    // clean up gpu
	    hipFree((void *) &dev_x);
	    hipFree((void *) &dev_y);
	    hipFree((void *) &dev_w);
	    hipFree((void *) &dev_sums_array);
	    hipFree((void *) &dev_norms);
	    hipFree((void *) &dev_blocks_norm);
	    hipFree((void *) &dev_y_new);

	    // clean up cpu
	    free(x);
	    free(y);
	    free(sums_array);

	    return 0;
}

__global__ void
multiply_arrays(double *x, double *w, double *y_new)
{
	double tmp = 0;
	
	int row, column;
	double dist;
	int global_thread_id = threadIdx.x + blockIdx.x * blockDim.x;

	int w_tid = (global_thread_id) * 3;
	while (w_tid < 3 * dev_sparse_count){
		row = w[w_tid];
		column = w[w_tid + 1];
		dist = w[w_tid + 2];

		// mulitply distance with every element in the row
		for(int i = 0; i < dev_columns; i++){
			tmp = dist * x[column * dev_columns + i];
			atomicAdd(&y_new[row * dev_columns + i], tmp);
		}
		w_tid += blockDim.x * gridDim.x * 3;
	}
}

__global__ void
get_sum_per_row(double *w, double *sums)
{
	int global_thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	int row;
	double dist;

	int w_tid = (global_thread_id) * 3;
	while (w_tid < dev_sparse_count * 3){
		row = w[w_tid];
		dist = w[w_tid + 2];
		atomicAdd(&sums[row], dist);
		
		w_tid += blockDim.x * gridDim.x * 3;
	}
}

__global__ void
mean_shift(double *y_new, double *sums_array, double *y, double *norms)
{
	int y_tid = threadIdx.x + blockIdx.x * blockDim.x;
	int sum_tid = y_tid / dev_columns;
	double m_dif = 0;
	int norms_tid = y_tid;
	norms[norms_tid] = 0;
	dev_sparse_count	= 0;

	while (y_tid < (dev_rows * dev_columns)){
		y_new[y_tid] /= sums_array[sum_tid];
		m_dif = y_new[y_tid] - y[y_tid];
		norms[norms_tid] += pow(m_dif, 2);
		y[y_tid] = y_new[y_tid];

		y_tid += gridDim.x * blockDim.x;
		sum_tid = y_tid / dev_columns;
	}
}

__global__ void 
compute_norms(double *norms, double *norm_per_block)
{
	int n_tid = 2 * (threadIdx.x + blockIdx.x * blockDim.x);
	int i = 1;
	int initial_tid = n_tid / 2;
	int block_limit = gridDim.x * blockDim.x;

	int block_end = 2 * (blockIdx.x * blockDim.x + blockDim.x) - 1; 

	if (n_tid < (2 * block_limit)){
		
		while ( (i < (2 * blockDim.x)) && n_tid < block_end && 
					 (n_tid + i) <= block_end){
			
			norms[n_tid] += norms[n_tid + i];
			n_tid = n_tid + i * (initial_tid * 2 - 2 * (blockIdx.x * blockDim.x));
			i *= 2;
			__syncthreads();
			
		}

		
		if (!((initial_tid) % blockDim.x))
			norm_per_block[blockIdx.x] = norms[n_tid];
		
	} 
}

__device__ double
compute_distance(double *y_i, double *x_j, double limit)
{
	double dist = 0, tmp;
	for (int i = 0; i < dev_columns; i++){
		tmp = y_i[i] - x_j[i];
		dist += tmp * tmp;
		if (dist > limit)
			return 0;
	}

	return exp( -dist / (2 * limit));
}

__global__ void
get_exp_shared(double *x, double *y, double *w)
{
	
	double dist;
	double limit = dev_h * dev_h;
	long int pos;
	
	__shared__ double s_y[BLOCK_SIZE][DIMS];
	__shared__ double s_x[BLOCK_SIZE][DIMS];
	
	/* upper_bound is used so the division with BLOCK_SIZE 
		 will give the bigger real mulitplier of BLOCK_SIZE	*/
	long int global_block_id = blockIdx.x;
	long int upper_bound = dev_rows + BLOCK_SIZE - 1;

	/* we	want block_row to change every rows elements
		 and block col every one BLOCK_SIZE elemetns		 */
	long int block_row = blockIdx.x / (upper_bound / BLOCK_SIZE);
	long int block_col = blockIdx.x % (upper_bound / BLOCK_SIZE);

	long int y_tid = threadIdx.x / (BLOCK_SIZE / stride) + block_row * BLOCK_SIZE;
	long int x_tid = stride * (threadIdx.x % (BLOCK_SIZE / stride)) + block_col * BLOCK_SIZE;

	long int upload_col = (x_tid - block_col * BLOCK_SIZE) % DIMS;
	long int upload_col_x = y_tid % DIMS;

	long int s_y_tid = y_tid - block_row * BLOCK_SIZE;
	long int s_x_tid = x_tid - block_col * BLOCK_SIZE;
	double loop_limit = (upper_bound * upper_bound ) / (BLOCK_SIZE * BLOCK_SIZE);
	
	int last_index = (upper_bound / BLOCK_SIZE - 1);

	int flag_row = ((block_col == last_index) && (dev_rows - BLOCK_SIZE * last_index) < DIMS);
	int flag_col = (block_row == last_index) && (dev_rows - BLOCK_SIZE * last_index) < DIMS;
	// TODO: if dims >> blocksize && not rows%dims == 0 problem!!!
	while (global_block_id < loop_limit){
		
		if (y_tid < dev_rows && x_tid < dev_rows){
			__syncthreads();
			
			if ((flag_row && x_tid == (dev_rows - stride)) || (x_tid == (((block_col+1)*BLOCK_SIZE) - 1)) && DIMS > BLOCK_SIZE){
				for (int i = 0; i < (DIMS - upload_col); i++)
					s_y[s_y_tid][upload_col + i] = y[y_tid * dev_columns + upload_col + i];
			}
			else
				for (int i = 0; (i < stride && (upload_col + i) < dev_columns); i++)
					s_y[s_y_tid][upload_col + i] = y[y_tid * dev_columns + upload_col + i];
			
			if ((flag_col && y_tid == (dev_rows - 1)) || (y_tid == (((block_row+1)*BLOCK_SIZE) - 1) && DIMS > BLOCK_SIZE))
			 for (int j = 0; j < (DIMS - upload_col_x); j++)
				 for (int i = 0; i < stride; i++)
					s_x[s_x_tid + i][upload_col_x + j] = x[(x_tid + i) * dev_columns + upload_col_x + j];
			else
				for (int i = 0; i < stride; i++)
					s_x[s_x_tid + i][upload_col_x] = x[(x_tid + i) * dev_columns + upload_col_x];
			__syncthreads();
			
			for (int i = 0; i < stride; i++){
				
				dist = compute_distance(&s_y[s_y_tid][0], &s_x[s_x_tid][0], limit);
			 
				if (dist){

					pos = atomicAdd(&dev_sparse_count, 1);
						
					w[pos * 3] = y_tid;
					w[pos * 3 + 1] = x_tid;
					w[pos * 3 + 2] = dist;
				}
				s_x_tid++;
				x_tid++;
			}
		}

		global_block_id += gridDim.x;

		block_row = global_block_id / (upper_bound / BLOCK_SIZE);
		block_col = global_block_id % (upper_bound / BLOCK_SIZE);

		y_tid = threadIdx.x / (BLOCK_SIZE / stride) + block_row * BLOCK_SIZE;
		x_tid = stride * (threadIdx.x % (BLOCK_SIZE / stride)) + block_col * BLOCK_SIZE;

		s_y_tid = y_tid - block_row * BLOCK_SIZE;
		s_x_tid = x_tid - block_col * BLOCK_SIZE;

		flag_row = (block_col == last_index) && (dev_rows - BLOCK_SIZE * last_index) < DIMS;
		flag_col = (block_row == last_index) && (dev_rows - BLOCK_SIZE * last_index) < DIMS;
	}

}

double *
read_data(const char *name)
{
	FILE *f;
	f = fopen(name, "rb");
	fseek(f, 0L, SEEK_END);
	int pos = ftell(f);
	fseek(f, 0L, SEEK_SET);

	int number_elements = pos / sizeof(double);
	double *x = (double *) malloc(sizeof *x * number_elements);
	fread(x, sizeof *x, number_elements, f);
	rows = number_elements / columns;
	fclose(f);

	return x;
}

int 
check_results(double *y, char *name)
{
	double *mat_res = read_data(name);
	double dist = 0;
	for (int i = 0; i < rows; i++){
		for (int j = 0; j < columns; j++){
			dist += pow(y[i * columns + j] - mat_res[i * columns + j], 2);
			// if (dist > 1)
					// printf("%f %f %f\n", y[i * columns + j], mat_res[i * columns + j], dist);
		}
	}
	printf("done dist %.10f \n", dist);
	return 0;
}

void
write_results(char **argv, int iterations, double time)
{
	FILE *f;
	f = fopen("results_shared.txt", "a");
	fprintf(f, "%d %d %s %d %f\n", rows, columns,argv[1], iterations, time);
	fclose(f);
}
